/***********************************************************************
/
/  GRID CLASS (RUNGE-KUTTA FIRST STEP ON GPU)
/
/  written by: Peng Wang
/  date:       September, 2012
/  modified1:
/
/
************************************************************************/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "ErrorExceptions.h"
#include "macros_and_parameters.h"
#include "typedefs.h"
#include "global_data.h"
#include "Fluxes.h"
#include "GridList.h"
#include "ExternalBoundary.h"
#include "TopGridData.h"
#include "Grid.h"
#include "CUDAUtil.h"


int CosmologyComputeExpansionFactor(FLOAT time, FLOAT *a, FLOAT *dadt);

int grid::CudaMHDRK2_1stStep(fluxes *SubgridFluxes[], 
                             int NumberOfSubgrids, int level,
                             ExternalBoundary *Exterior)
  /*
    NumberOfSubgrids: the actual number of subgrids + 1
    SubgridFluxes[NumberOfSubgrids]
  */
{
  if (ProcessorNumber != MyProcessorNumber) {
    return SUCCESS;
  }

  if (NumberOfBaryonFields == 0) {
    return SUCCESS;
  }

  int size = 1;
  for (int dim = 0; dim < GridRank; dim++)
    size *= GridDimension[dim];

  /* allocate space for fluxes */
  int fluxsize;
  for (int subgrid = 0; subgrid < NumberOfSubgrids; subgrid++) {
    for (int flux = 0; flux < GridRank; flux++)  {
      
      fluxsize = 1;
      for (int j = 0; j < GridRank; j++) {
	fluxsize *= SubgridFluxes[subgrid]->LeftFluxEndGlobalIndex[flux][j] -
	  SubgridFluxes[subgrid]->LeftFluxStartGlobalIndex[flux][j] + 1;
      }
      
      for (int j = GridRank; j < 3; j++) {
	SubgridFluxes[subgrid]->LeftFluxStartGlobalIndex[flux][j] = 0;
	SubgridFluxes[subgrid]->LeftFluxEndGlobalIndex[flux][j] = 0;
	SubgridFluxes[subgrid]->RightFluxStartGlobalIndex[flux][j] = 0;
	SubgridFluxes[subgrid]->RightFluxEndGlobalIndex[flux][j] = 0;
      }
       
      for (int field = 0; field < NumberOfBaryonFields; field++) {
	if (SubgridFluxes[subgrid]->LeftFluxes[field][flux] == NULL) {
	  SubgridFluxes[subgrid]->LeftFluxes[field][flux]  = new float[fluxsize];
	}
	if (SubgridFluxes[subgrid]->RightFluxes[field][flux] == NULL)
	  SubgridFluxes[subgrid]->RightFluxes[field][flux] = new float[fluxsize];
	for (int n = 0; n < fluxsize; n++) {
	  SubgridFluxes[subgrid]->LeftFluxes[field][flux][n] = 0.0;
	  SubgridFluxes[subgrid]->RightFluxes[field][flux][n] = 0.0;
	}
      }
      
      for (int field = NumberOfBaryonFields; field < MAX_NUMBER_OF_BARYON_FIELDS; field++) {
	SubgridFluxes[subgrid]->LeftFluxes[field][flux] = NULL;
	SubgridFluxes[subgrid]->RightFluxes[field][flux] = NULL;
      }
      
    }  // next flux
    
    for (int flux = GridRank; flux < 3; flux++) {
      for (int field = 0; field < MAX_NUMBER_OF_BARYON_FIELDS; field++) {
	SubgridFluxes[subgrid]->LeftFluxes[field][flux] = NULL;
	SubgridFluxes[subgrid]->RightFluxes[field][flux] = NULL;
      }
    }
    
  } // end of loop over subgrids

  /* RK2 first step */

  int DensNum, GENum, TENum, Vel1Num, Vel2Num, Vel3Num;
  int B1Num, B2Num, B3Num, PhiNum;
  this->IdentifyPhysicalQuantities(DensNum, GENum, Vel1Num, Vel2Num, 
                                   Vel3Num, TENum, B1Num, B2Num, B3Num, 
                                   PhiNum);
  int DeNum, HINum, HIINum, HeINum, HeIINum, HeIIINum, HMNum, H2INum, H2IINum,
      DINum, DIINum, HDINum;
  if (MultiSpecies)
    this->IdentifySpeciesFields(DeNum, HINum, HIINum, HeINum, HeIINum, HeIIINum, 
                                HMNum, H2INum, H2IINum, DINum, DIINum, HDINum);

  const size_t sizebytes = size*sizeof(float);
  //
  // Allocate memory on GPU
  //
  this->CudaMHDMallocGPUData();

  //
  // Copy data from CPU to GPU
  //
  if (SelfGravity || ExternalGravity || UniformGravity || PointSourceGravity) 
    for (int i = 0; i < GridRank; i++)
      hipMemcpy(MHDData.AccelerationField[i], AccelerationField[i], sizebytes, hipMemcpyHostToDevice);
  if (UseDrivingField) {
    int Drive1Num, Drive2Num, Drive3Num;
    if (IdentifyDrivingFields(Drive1Num, Drive2Num, Drive3Num) == FAIL) {
      printf("grid::CudaMHDRK2_1stStep: canot identify driving fields.\n");
      return FAIL;
    }
    hipMemcpy(MHDData.DrivingForce[0], BaryonField[Drive1Num], sizebytes, hipMemcpyHostToDevice);
    hipMemcpy(MHDData.DrivingForce[1], BaryonField[Drive2Num], sizebytes, hipMemcpyHostToDevice);
    hipMemcpy(MHDData.DrivingForce[2], BaryonField[Drive3Num], sizebytes, hipMemcpyHostToDevice);
  }   
  hipMemcpy(MHDData.D  , BaryonField[DensNum], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.V1 , BaryonField[Vel1Num], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.V2 , BaryonField[Vel2Num], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.V3 , BaryonField[Vel3Num], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.TE , BaryonField[TENum  ], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.B1 , BaryonField[B1Num  ], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.B2 , BaryonField[B2Num  ], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.B3 , BaryonField[B3Num  ], sizebytes, hipMemcpyHostToDevice);
  hipMemcpy(MHDData.Phi, BaryonField[PhiNum ], sizebytes, hipMemcpyHostToDevice);

  // copy to old baryon
  hipMemcpy(MHDData.OldD  , MHDData.D  , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldV1 , MHDData.V1 , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldV2 , MHDData.V2 , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldV3 , MHDData.V3 , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldTE , MHDData.TE , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldB1 , MHDData.B1 , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldB2 , MHDData.B2 , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldB3 , MHDData.B3 , sizebytes, hipMemcpyDeviceToDevice);
  hipMemcpy(MHDData.OldPhi, MHDData.Phi, sizebytes, hipMemcpyDeviceToDevice);

  if (MultiSpecies) {
    hipMemcpy(MHDData.Species[0], BaryonField[HINum], sizebytes, hipMemcpyHostToDevice);
    hipMemcpy(MHDData.Species[1], BaryonField[HIINum], sizebytes, hipMemcpyHostToDevice);
    hipMemcpy(MHDData.Species[2], BaryonField[HeINum], sizebytes, hipMemcpyHostToDevice);
    hipMemcpy(MHDData.Species[3], BaryonField[HeIINum], sizebytes, hipMemcpyHostToDevice);
    hipMemcpy(MHDData.Species[4], BaryonField[HeIIINum], sizebytes, hipMemcpyHostToDevice);
    if (MultiSpecies > 1) {
      hipMemcpy(MHDData.Species[5], BaryonField[HMNum], sizebytes, hipMemcpyHostToDevice);
      hipMemcpy(MHDData.Species[6], BaryonField[H2INum], sizebytes, hipMemcpyHostToDevice);
      hipMemcpy(MHDData.Species[7], BaryonField[H2IINum], sizebytes, hipMemcpyHostToDevice);
    }
    if (MultiSpecies > 2) {
      hipMemcpy(MHDData.Species[8], BaryonField[DINum], sizebytes, hipMemcpyHostToDevice);
      hipMemcpy(MHDData.Species[9], BaryonField[DIINum], sizebytes, hipMemcpyHostToDevice);
      hipMemcpy(MHDData.Species[10], BaryonField[HDINum], sizebytes, hipMemcpyHostToDevice);
    }
    // copy to old species
    for (int i = 0; i < NSpecies; i++)
      hipMemcpy(MHDData.OldSpecies[i], MHDData.Species[i], sizebytes,
                 hipMemcpyDeviceToDevice);
  }
  CUDA_SAFE_CALL( hipGetLastError() );
  //
  // Solve MHD equations on GPU
  //
  this->CudaSolveMHDEquations(SubgridFluxes, NumberOfSubgrids, 1);

  //
  // Copy results from CPU to GPU                                                    
  //
  hipMemcpy(BaryonField[DensNum], MHDData.D  , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[Vel1Num], MHDData.V1 , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[Vel2Num], MHDData.V2 , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[Vel3Num], MHDData.V3 , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[TENum  ], MHDData.TE , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[B1Num  ], MHDData.B1 , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[B2Num  ], MHDData.B2 , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[B3Num  ], MHDData.B3 , sizebytes, hipMemcpyDeviceToHost);
  hipMemcpy(BaryonField[PhiNum ], MHDData.Phi, sizebytes, hipMemcpyDeviceToHost);
  if (MultiSpecies) {
    hipMemcpy(BaryonField[HINum   ], MHDData.Species[0], sizebytes, hipMemcpyDeviceToHost);
    hipMemcpy(BaryonField[HIINum  ], MHDData.Species[1], sizebytes, hipMemcpyDeviceToHost);
    hipMemcpy(BaryonField[HeINum  ], MHDData.Species[2], sizebytes, hipMemcpyDeviceToHost);
    hipMemcpy(BaryonField[HeIINum ], MHDData.Species[3], sizebytes, hipMemcpyDeviceToHost);
    hipMemcpy(BaryonField[HeIIINum], MHDData.Species[4], sizebytes, hipMemcpyDeviceToHost);
    if (MultiSpecies > 1) {
      hipMemcpy(BaryonField[HMNum  ], MHDData.Species[5], sizebytes, hipMemcpyDeviceToHost);
      hipMemcpy(BaryonField[H2INum ], MHDData.Species[6], sizebytes, hipMemcpyDeviceToHost);
      hipMemcpy(BaryonField[H2IINum], MHDData.Species[7], sizebytes, hipMemcpyDeviceToHost);
    }
    if (MultiSpecies > 2) {
      hipMemcpy(BaryonField[DINum ], MHDData.Species[8], sizebytes, hipMemcpyDeviceToHost);
      hipMemcpy(BaryonField[DIINum], MHDData.Species[9], sizebytes, hipMemcpyDeviceToHost);
      hipMemcpy(BaryonField[HDINum], MHDData.Species[10],sizebytes, hipMemcpyDeviceToHost);
    }
  }
  if (NSpecies > 0)
    this->UpdateElectronDensity();

  return SUCCESS;
}

